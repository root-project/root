/*
 * Project: RooFit
 * Author:
 *   Jonas Rembser, CERN 2023
 *
 * Copyright (c) 2023, CERN
 *
 * Redistribution and use in source and binary forms,
 * with or without modification, are permitted according to the terms
 * listed in LICENSE (http://roofit.sourceforge.net/license.txt)
 */

#include <RooFit/Detail/CudaInterface.h>

#include <stdexcept>
#include <sstream>
#include <string>

#define ERRCHECK(err) __checkCudaErrors((err), __func__, __FILE__, __LINE__)
inline static void __checkCudaErrors(hipError_t error, std::string func, std::string file, int line)
{
   if (error != hipSuccess) {
      std::stringstream errMsg;
      errMsg << func << "(), " << file + ":" << std::to_string(line) << " : " << hipGetErrorString(error);
      throw std::runtime_error(errMsg.str());
   }
}

namespace RooFit {
namespace Detail {
namespace CudaInterface {

DeviceMemory::DeviceMemory(std::size_t n, std::size_t typeSize) : _size{n}
{
   void *ret;
   ERRCHECK(::hipMalloc(&ret, n * typeSize));
   _data.reset(ret);
}
PinnedHostMemory::PinnedHostMemory(std::size_t n, std::size_t typeSize) : _size{n}
{
   void *ret;
   ERRCHECK(::hipHostMalloc(&ret, n * typeSize));
   _data.reset(ret);
}

template <>
void Deleter<DeviceMemory>::operator()(void *ptr)
{
   ERRCHECK(::hipFree(ptr));
   ptr = nullptr;
}
template <>
void Deleter<PinnedHostMemory>::operator()(void *ptr)
{
   ERRCHECK(::hipHostFree(ptr));
   ptr = nullptr;
}

/**
 * Creates a new CUDA event.
 *
 * @param[in] forTiming       Set to true if the event is intended for timing purposes.
 *                            If `false`, the `hipEventDisableTiming` is passed to CUDA.
 * @return                    CudaEvent object representing the new event.
 */
CudaEvent::CudaEvent(bool forTiming)
{
   auto event = new hipEvent_t;
   ERRCHECK(hipEventCreateWithFlags(event, forTiming ? 0 : hipEventDisableTiming));
   _ptr.reset(event);
}

template <>
void Deleter<CudaEvent>::operator()(void *ptr)
{
   auto event = reinterpret_cast<hipEvent_t *>(ptr);
   ERRCHECK(hipEventDestroy(*event));
   delete event;
   ptr = nullptr;
}

template <>
void Deleter<CudaStream>::operator()(void *ptr)
{
   auto stream = reinterpret_cast<hipStream_t *>(ptr);
   ERRCHECK(hipStreamDestroy(*stream));
   delete stream;
   ptr = nullptr;
}

/**
 * Records a CUDA event.
 *
 * @param[in] event           CudaEvent object representing the event to be recorded.
 * @param[in] stream          CudaStream in which to record the event.
 */
void hipEventRecord(CudaEvent &event, CudaStream &stream)
{
   ERRCHECK(::hipEventRecord(event, stream));
}

/**
 * Creates a new CUDA stream.
 *
 * @return                    CudaStream object representing the new stream.
 */
CudaStream::CudaStream()
{
   auto stream = new hipStream_t;
   ERRCHECK(hipStreamCreate(stream));
   _ptr.reset(stream);
}

/**
 * Checks if a CUDA stream is currently active.
 *
 * @return                    True if the stream is active, false otherwise.
 */
bool CudaStream::isActive()
{
   hipError_t err = hipStreamQuery(*this);
   if (err == hipErrorNotReady)
      return true;
   else if (err == hipSuccess)
      return false;
   ERRCHECK(err);
   return false;
}

/**
 * Makes a CUDA stream wait for a CUDA event.
 *
 * @param[in] event           CudaEvent object representing the event to wait for.
 */
void CudaStream::waitForEvent(CudaEvent &event)
{
   ERRCHECK(::hipStreamWaitEvent(*this, event, 0));
}

/**
 * Calculates the elapsed time between two CUDA events.
 *
 * @param[in] begin           CudaEvent representing the start event.
 * @param[in] end             CudaEvent representing the end event.
 * @return                    Elapsed time in milliseconds.
 */
float hipEventElapsedTime(CudaEvent &begin, CudaEvent &end)
{
   float ret;
   ERRCHECK(::hipEventElapsedTime(&ret, begin, end));
   return ret;
}

/// \cond ROOFIT_INTERNAL

void copyHostToDeviceImpl(const void *src, void *dest, size_t nBytes, CudaStream *stream)
{
   if (stream)
      ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyHostToDevice, *stream));
   else
      ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyHostToDevice));
}

void copyDeviceToHostImpl(const void *src, void *dest, size_t nBytes, CudaStream *stream)
{
   if (stream)
      ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyDeviceToHost, *stream));
   else
      ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToHost));
}

/// \endcond

} // namespace CudaInterface
} // namespace Detail
} // namespace RooFit
