/*
 * Project: RooFit
 * Authors:
 *   Emmanouil Michalainas, CERN, September 2020
 *
 * Copyright (c) 2021, CERN
 *
 * Redistribution and use in source and binary forms,
 * with or without modification, are permitted according to the terms
 * listed in LICENSE (http://roofit.sourceforge.net/license.txt)
 */

/**
\file RooBatchCompute.cu
\class RbcClass
\ingroup Roobatchcompute

This file contains the code for cuda computations using the RooBatchCompute library.
**/

#include "RooBatchCompute.h"
#include "Batches.h"

#include "ROOT/RConfig.h"
#include "TError.h"

#include <algorithm>
#include <thrust/reduce.h>

#ifdef __HIPCC__
#define ERRCHECK(err) __checkCudaErrors((err), __func__, __FILE__, __LINE__)
inline static void __checkCudaErrors(hipError_t error, std::string func, std::string file, int line)
{
   if (error != hipSuccess) {
      Fatal((func + "(), " + file + ":" + std::to_string(line)).c_str(), "%s", hipGetErrorString(error));
      throw std::bad_alloc();
   }
}
#endif

#ifndef RF_ARCH
#error "RF_ARCH should always be defined"
#endif

namespace RooBatchCompute {
namespace RF_ARCH {

std::vector<void (*)(Batches)> getFunctions();

/// This class overrides some RooBatchComputeInterface functions, for the
/// purpose of providing a cuda specific implementation of the library.
class RooBatchComputeClass : public RooBatchComputeInterface {
private:
   const std::vector<void (*)(Batches)> _computeFunctions;

public:
   RooBatchComputeClass() : _computeFunctions(getFunctions())
   {
      dispatchCUDA = this; // Set the dispatch pointer to this instance of the library upon loading
   }

   Architecture architecture() const override { return Architecture::RF_ARCH; };
   std::string architectureName() const override
   {
      // transform to lower case to match the original architecture name passed to the compiler
      std::string out = _QUOTE_(RF_ARCH);
      std::transform(out.begin(), out.end(), out.begin(), [](unsigned char c) { return std::tolower(c); });
      ;
      return out;
   };

   /** Compute multiple values using cuda kernels.
   This method creates a Batches object and passes it to the correct compute function.
   The compute function is launched as a cuda kernel.
   \param computer An enum specifying the compute function to be used.
   \param output The array where the computation results are stored.
   \param nEvents The number of events to be processed.
   \param vars A std::vector containing pointers to the variables involved in the computation.
   \param extraArgs An optional std::vector containing extra double values that may participate in the computation. **/
   void compute(hipStream_t *stream, Computer computer, RestrictArr output, size_t nEvents, const VarVector &vars,
                const ArgVector &extraArgs) override
   {
      Batches batches(output, nEvents, vars, extraArgs);
      _computeFunctions[computer]<<<128, 512, 0, *stream>>>(batches);
   }
   /// Return the sum of an input array
   double sumReduce(hipStream_t *stream, InputArr input, size_t n) override
   {
      return thrust::reduce(thrust::cuda::par.on(*stream), input, input + n, 0.0);
   }

   // cuda functions
   virtual void *hipMalloc(size_t nBytes)
   {
      void *ret;
      ERRCHECK(::hipMalloc(&ret, nBytes));
      return ret;
   }
   virtual void hipFree(void *ptr) { ERRCHECK(::hipFree(ptr)); }
   virtual void *hipHostMalloc(size_t nBytes)
   {
      void *ret;
      ERRCHECK(::hipHostMalloc(&ret, nBytes));
      return ret;
   }
   virtual void hipHostFree(void *ptr) { ERRCHECK(::hipHostFree(ptr)); }
   virtual hipEvent_t *newCudaEvent(bool forTiming)
   {
      auto ret = new hipEvent_t;
      ERRCHECK(hipEventCreateWithFlags(ret, forTiming ? 0 : hipEventDisableTiming));
      return ret;
   }
   virtual void deleteCudaEvent(hipEvent_t *event)
   {
      ERRCHECK(hipEventDestroy(*event));
      delete event;
   }
   virtual void hipEventRecord(hipEvent_t *event, hipStream_t *stream)
   {
      ERRCHECK(::hipEventRecord(*event, *stream));
   }
   virtual hipStream_t *newCudaStream()
   {
      auto ret = new hipStream_t;
      ERRCHECK(hipStreamCreate(ret));
      return ret;
   }
   virtual void deleteCudaStream(hipStream_t *stream)
   {
      ERRCHECK(hipStreamDestroy(*stream));
      delete stream;
   }
   virtual bool streamIsActive(hipStream_t *stream)
   {
      hipError_t err = hipStreamQuery(*stream);
      if (err == hipErrorNotReady)
         return true;
      else if (err == hipSuccess)
         return false;
      ERRCHECK(err);
      return false;
   }
   virtual void hipStreamWaitEvent(hipStream_t *stream, hipEvent_t *event)
   {
      ERRCHECK(::hipStreamWaitEvent(*stream, *event, 0));
   }
   virtual float hipEventElapsedTime(hipEvent_t *begin, hipEvent_t *end)
   {
      float ret;
      ERRCHECK(::hipEventElapsedTime(&ret, *begin, *end));
      return ret;
   }
   void memcpyToCUDA(void *dest, const void *src, size_t nBytes, hipStream_t *stream) override
   {
      if (stream)
         ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyHostToDevice, *stream));
      else
         ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyHostToDevice));
   }
   void memcpyToCPU(void *dest, const void *src, size_t nBytes, hipStream_t *stream) override
   {
      if (stream)
         ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyDeviceToHost, *stream));
      else
         ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToHost));
   }
}; // End class RooBatchComputeClass

/// Static object to trigger the constructor which overwrites the dispatch pointer.
static RooBatchComputeClass computeObj;

/** Construct a Batches object
\param output The array where the computation results are stored.
\param nEvents The number of events to be processed.
\param vars A std::vector containing pointers to the variables involved in the computation.
\param extraArgs An optional std::vector containing extra double values that may participate in the computation.
For every scalar parameter a `Batch` object inside the `Batches` object is set accordingly;
a data member of type double gets assigned the scalar value. This way, when the cuda kernel
is launched this scalar value gets copied automatically and thus no call to hipMemcpy is needed **/
Batches::Batches(RestrictArr output, size_t nEvents, const VarVector &vars, const ArgVector &extraArgs, double *)
   : _nEvents(nEvents), _nBatches(vars.size()), _nExtraArgs(extraArgs.size()), _output(output)
{
   if (vars.size() > maxParams) {
      throw std::runtime_error(std::string("Size of vars is ") + std::to_string(vars.size()) +
                               ", which is larger than maxParams = " + std::to_string(maxParams) + "!");
   }
   if (extraArgs.size() > maxExtraArgs) {
      throw std::runtime_error(std::string("Size of extraArgs is ") + std::to_string(extraArgs.size()) +
                               ", which is larger than maxExtraArgs = " + std::to_string(maxExtraArgs) + "!");
   }

   for (int i = 0; i < vars.size(); i++) {
      const RooSpan<const double> &span = vars[i];
      size_t size = span.size();
      if (size == 1)
         _arrays[i].set(span[0], nullptr, false);
      else
         _arrays[i].set(0.0, span.data(), true);
   }
   std::copy(extraArgs.cbegin(), extraArgs.cend(), _extraArgs);
}

} // End namespace RF_ARCH
} // End namespace RooBatchCompute
