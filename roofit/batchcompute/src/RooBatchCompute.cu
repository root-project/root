// rbc library created September 2020 by Emmanouil Michalainas

#include "rbc.h"
#include "Batches.h"

#include "TError.h"

#include <thrust/reduce.h>

namespace rbc {
namespace RF_ARCH {
  
std::vector<void(*)(Batches)> getFunctions();

class RbcClass : public RbcInterface {
  private:
    const std::vector<void(*)(Batches)> _computeFunctions;
  public:
    RbcClass() : _computeFunctions(getFunctions())
    {
      dispatchCUDA = this; // Set the dispatch pointer to this instance of the library upon loading
    }

    void init()
    {
      hipError_t err = hipSetDevice(0);
      if (err==hipSuccess) hipFree(nullptr);
      else
      {
        dispatchCUDA = nullptr;
        Error( (std::string(__func__)+"(), "+__FILE__+":"+std::to_string(__LINE__)).c_str(), "%s", hipGetErrorString(err) );
      }
    }

    void compute(Computer computer, RestrictArr output, size_t nEvents, const DataMap& varData, const VarVector& vars, const ArgVector& extraArgs) override  
    {
      Batches batches(output, nEvents, varData, vars, extraArgs);
      _computeFunctions[computer]<<<128,512>>>(batches);
    }
    double sumReduce(InputArr input, size_t n) override {
      return thrust::reduce(thrust::device, input, input+n, 0.0);
    }

    //cuda functions
    virtual void* hipMalloc(size_t nBytes) {
      void* ret;
      ERRCHECK( ::hipMalloc(&ret, nBytes) );
      return ret;
    }
    virtual void hipFree(void* ptr) {
      ERRCHECK( ::hipFree(ptr) );
    }
    virtual void* hipHostMalloc(size_t nBytes) {
      void* ret;
      ERRCHECK( ::hipHostMalloc(&ret, nBytes) );
      return ret;
    }
    virtual void hipHostFree(void* ptr) {
      ERRCHECK( ::hipHostFree(ptr) );
    }
    virtual hipEvent_t* newCudaEvent(bool forTiming) {
      auto ret = new hipEvent_t;
      ERRCHECK( hipEventCreateWithFlags(ret, forTiming ? 0 : hipEventDisableTiming) );
      return ret;
    }
    virtual void deleteCudaEvent(hipEvent_t* event) {
      ERRCHECK( hipEventDestroy(*event) );
      delete event;
    }
    virtual void hipEventRecord(hipEvent_t* event, hipStream_t* stream) {
      ERRCHECK( ::hipEventRecord(*event, *stream) );
    }
    virtual hipStream_t* newCudaStream() {
      auto ret = new hipStream_t;
      ERRCHECK( hipStreamCreate(ret) );
      return ret;
    }
    virtual void deleteCudaStream(hipStream_t* stream) {
      ERRCHECK( hipStreamDestroy(*stream) );
      delete stream;
    }
    virtual bool streamIsActive(hipStream_t* stream) {
      hipError_t err = hipStreamQuery(*stream);
      if (err==hipErrorNotReady) return true;
      else if (err==hipSuccess) return false;
      ERRCHECK(err);
      return false;
    }
    virtual void hipStreamWaitEvent(hipStream_t* stream, hipEvent_t* event) {
      ERRCHECK( ::hipStreamWaitEvent(*stream, *event) );
    }
    virtual float hipEventElapsedTime(hipEvent_t* begin, hipEvent_t* end) {
      float ret;
      ERRCHECK( ::hipEventElapsedTime(&ret, *begin, *end) );
      return ret;
    }
    void memcpyToCUDA(void* dest, const void* src, size_t nBytes, hipStream_t* stream) override {
      if (stream)
        ERRCHECK( hipMemcpyAsync(dest, src, nBytes, hipMemcpyHostToDevice, *stream) );
      else
        ERRCHECK( hipMemcpy(dest, src, nBytes, hipMemcpyHostToDevice) );
    }
    void memcpyToCPU(void* dest, const void* src, size_t nBytes, hipStream_t* stream) override {
      if (stream)
        ERRCHECK( hipMemcpyAsync(dest, src, nBytes, hipMemcpyDeviceToHost, *stream) );
      else
        ERRCHECK( hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToHost) );
    }
}; // End class RbcClass

/// Static object to trigger the constructor which overwrites the dispatch pointer.
static RbcClass computeObj;


Batches::Batches(RestrictArr output, size_t nEvents, const DataMap& varData, const VarVector& vars, const ArgVector& extraArgs, double[maxParams][bufferSize])
  : _nEvents(nEvents), _nBatches(vars.size()), _nExtraArgs(extraArgs.size()), _output(output)
{  
  for (int i=0; i<vars.size(); i++)
  {
    const RooSpan<const double>& span = varData.at(vars[i]);
    size_t size = span.size();
    if (size==1) _arrays[i].set(span[0], nullptr, false);
    else _arrays[i].set(0.0, span.data(), true);
  }
  std::copy(extraArgs.cbegin(), extraArgs.cend(), _extraArgs);
}

} // End namespace RF_ARCH
} // End namespace rbc
