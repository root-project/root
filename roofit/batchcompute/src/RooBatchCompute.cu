#include "hip/hip_runtime.h"
/*
 * Project: RooFit
 * Authors:
 *   Emmanouil Michalainas, CERN, September 2020
 *
 * Copyright (c) 2021, CERN
 *
 * Redistribution and use in source and binary forms,
 * with or without modification, are permitted according to the terms
 * listed in LICENSE (http://roofit.sourceforge.net/license.txt)
 */

/**
\file RooBatchCompute.cu
\class RbcClass
\ingroup Roobatchcompute

This file contains the code for cuda computations using the RooBatchCompute library.
**/

#include "RooBatchCompute.h"
#include "Batches.h"

#include "ROOT/RConfig.hxx"
#include "TError.h"

#include <algorithm>

#ifdef __HIPCC__
#define ERRCHECK(err) __checkCudaErrors((err), __func__, __FILE__, __LINE__)
inline static void __checkCudaErrors(hipError_t error, std::string func, std::string file, int line)
{
   if (error != hipSuccess) {
      Fatal((func + "(), " + file + ":" + std::to_string(line)).c_str(), "%s", hipGetErrorString(error));
      throw std::bad_alloc();
   }
}
#endif

#ifndef RF_ARCH
#error "RF_ARCH should always be defined"
#endif

namespace RooBatchCompute {
namespace RF_ARCH {

constexpr int gridSize = 128;
constexpr int blockSize = 512;

std::vector<void (*)(Batches)> getFunctions();

/// This class overrides some RooBatchComputeInterface functions, for the
/// purpose of providing a cuda specific implementation of the library.
class RooBatchComputeClass : public RooBatchComputeInterface {
private:
   const std::vector<void (*)(Batches)> _computeFunctions;

public:
   RooBatchComputeClass() : _computeFunctions(getFunctions())
   {
      dispatchCUDA = this; // Set the dispatch pointer to this instance of the library upon loading
   }

   Architecture architecture() const override { return Architecture::RF_ARCH; };
   std::string architectureName() const override
   {
      // transform to lower case to match the original architecture name passed to the compiler
      std::string out = _QUOTE_(RF_ARCH);
      std::transform(out.begin(), out.end(), out.begin(), [](unsigned char c) { return std::tolower(c); });
      ;
      return out;
   };

   /** Compute multiple values using cuda kernels.
   This method creates a Batches object and passes it to the correct compute function.
   The compute function is launched as a cuda kernel.
   \param computer An enum specifying the compute function to be used.
   \param output The array where the computation results are stored.
   \param nEvents The number of events to be processed.
   \param vars A std::vector containing pointers to the variables involved in the computation.
   \param extraArgs An optional std::vector containing extra double values that may participate in the computation. **/
   void compute(hipStream_t *stream, Computer computer, RestrictArr output, size_t nEvents, const VarVector &vars,
                ArgVector &extraArgs) override
   {
      Batches batches(output, nEvents, vars, extraArgs);
      _computeFunctions[computer]<<<gridSize, blockSize, 0, *stream>>>(batches);
   }
   /// Return the sum of an input array
   double reduceSum(hipStream_t *stream, InputArr input, size_t n) override;
   ReduceNLLOutput reduceNLL(hipStream_t *, RooSpan<const double> probas, RooSpan<const double> weightSpan,
                             RooSpan<const double> weights, double weightSum,
                             RooSpan<const double> binVolumes) override;

   // cuda functions
   virtual void *hipMalloc(size_t nBytes)
   {
      void *ret;
      ERRCHECK(::hipMalloc(&ret, nBytes));
      return ret;
   }
   virtual void hipFree(void *ptr) { ERRCHECK(::hipFree(ptr)); }
   virtual void *hipHostMalloc(size_t nBytes)
   {
      void *ret;
      ERRCHECK(::hipHostMalloc(&ret, nBytes));
      return ret;
   }
   virtual void hipHostFree(void *ptr) { ERRCHECK(::hipHostFree(ptr)); }
   virtual hipEvent_t *newCudaEvent(bool forTiming)
   {
      auto ret = new hipEvent_t;
      ERRCHECK(hipEventCreateWithFlags(ret, forTiming ? 0 : hipEventDisableTiming));
      return ret;
   }
   virtual void deleteCudaEvent(hipEvent_t *event)
   {
      ERRCHECK(hipEventDestroy(*event));
      delete event;
   }
   virtual void hipEventRecord(hipEvent_t *event, hipStream_t *stream)
   {
      ERRCHECK(::hipEventRecord(*event, *stream));
   }
   virtual hipStream_t *newCudaStream()
   {
      auto ret = new hipStream_t;
      ERRCHECK(hipStreamCreate(ret));
      return ret;
   }
   virtual void deleteCudaStream(hipStream_t *stream)
   {
      ERRCHECK(hipStreamDestroy(*stream));
      delete stream;
   }
   virtual bool streamIsActive(hipStream_t *stream)
   {
      hipError_t err = hipStreamQuery(*stream);
      if (err == hipErrorNotReady)
         return true;
      else if (err == hipSuccess)
         return false;
      ERRCHECK(err);
      return false;
   }
   virtual void hipStreamWaitEvent(hipStream_t *stream, hipEvent_t *event)
   {
      ERRCHECK(::hipStreamWaitEvent(*stream, *event, 0));
   }
   virtual float hipEventElapsedTime(hipEvent_t *begin, hipEvent_t *end)
   {
      float ret;
      ERRCHECK(::hipEventElapsedTime(&ret, *begin, *end));
      return ret;
   }
   void memcpyToCUDA(void *dest, const void *src, size_t nBytes, hipStream_t *stream) override
   {
      if (stream)
         ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyHostToDevice, *stream));
      else
         ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyHostToDevice));
   }
   void memcpyToCPU(void *dest, const void *src, size_t nBytes, hipStream_t *stream) override
   {
      if (stream)
         ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyDeviceToHost, *stream));
      else
         ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToHost));
   }
}; // End class RooBatchComputeClass

template <class T>
class DeviceArray {
public:
   DeviceArray(std::size_t n) : _size{n} { hipMalloc(reinterpret_cast<void **>(&_deviceArray), n * sizeof(T)); }
   DeviceArray(T const *hostArray, std::size_t n) : _size{n}
   {
      hipMalloc((void **)&_deviceArray, n * sizeof(T));
      hipMemcpy(_deviceArray, hostArray, n * sizeof(T), hipMemcpyHostToDevice);
   }
   DeviceArray(DeviceArray const &other) = delete;
   DeviceArray &operator=(DeviceArray const &other) = delete;
   ~DeviceArray() { hipFree(_deviceArray); }

   std::size_t size() const { return _size; }
   T *data() { return _deviceArray; }
   T const *data() const { return _deviceArray; }

   void copyBack(T *hostArray, std::size_t n)
   {
      hipMemcpy(hostArray, _deviceArray, sizeof(T) * n, hipMemcpyDeviceToHost);
   }

private:
   T *_deviceArray = nullptr;
   std::size_t _size = 0;
};

template <class T, class U>
__global__ void sumMultiBlock(const T *__restrict__ gArr, int arraySize, U *__restrict__ gOut)
{
   int thIdx = threadIdx.x;
   int gthIdx = thIdx + blockIdx.x * blockSize;
   const int gridSize = blockSize * gridDim.x;
   U sum = 0;
   for (int i = gthIdx; i < arraySize; i += gridSize)
      sum += gArr[i];
   __shared__ U shArr[blockSize];
   shArr[thIdx] = sum;
   __syncthreads();
   for (int size = blockSize / 2; size > 0; size /= 2) { // uniform
      if (thIdx < size)
         shArr[thIdx] += shArr[thIdx + size];
      __syncthreads();
   }
   if (thIdx == 0)
      gOut[blockIdx.x] = shArr[0];
}

__global__ void nllSumMultiBlock(const double *__restrict__ probas, int probasSize, double *__restrict__ out)
{
   int thIdx = threadIdx.x;
   int gthIdx = thIdx + blockIdx.x * blockSize;
   const int gridSize = blockSize * gridDim.x;
   double sum = 0;
   for (int i = gthIdx; i < probasSize; i += gridSize)
      sum -= std::log(probas[i]);
   __shared__ double shArr[blockSize];
   shArr[thIdx] = sum;
   __syncthreads();
   for (int size = blockSize / 2; size > 0; size /= 2) { // uniform
      if (thIdx < size)
         shArr[thIdx] += shArr[thIdx + size];
      __syncthreads();
   }
   if (thIdx == 0)
      out[blockIdx.x] = shArr[0];
}

__global__ void nllSumKernel(const double *probas, double *out, int n)
{
   int idx = threadIdx.x;
   double nllSum = 0;
   for (int i = idx; i < n; i += blockSize) {
      nllSum -= std::log(probas[i]);
   }
   __shared__ double r[blockSize];
   r[idx] = nllSum;
   __syncthreads();
   for (int size = blockSize / 2; size > 0; size /= 2) { // uniform
      if (idx < size) {
         r[idx] += r[idx + size];
      }
      __syncthreads();
   }
   if (idx == 0) {
      *out = r[0];
   }
}

__global__ void nllSumWeightedKernel(const double *probas, const double *weightSpan, double *out, int n)
{
   int idx = threadIdx.x;
   double nllSum = 0;
   for (int i = idx; i < n; i += blockSize) {
      if (weightSpan[i] != 0.0) {
         nllSum -= weightSpan[i] * std::log(probas[i]);
      }
   }
   __shared__ double r[blockSize];
   r[idx] = nllSum;
   __syncthreads();
   for (int size = blockSize / 2; size > 0; size /= 2) { // uniform
      if (idx < size) {
         r[idx] += r[idx + size];
      }
      __syncthreads();
   }
   if (idx == 0) {
      *out = r[0];
   }
}

double RooBatchComputeClass::reduceSum(hipStream_t *stream, InputArr input, size_t n)
{
   DeviceArray<double> devOut{gridSize};
   double tmp = 0.0;
   sumMultiBlock<<<gridSize, blockSize, 0, *stream>>>(input, n, devOut.data());
   sumMultiBlock<<<1, blockSize, 0, *stream>>>(devOut.data(), gridSize, devOut.data());
   devOut.copyBack(&tmp, 1);
   return tmp;
}

ReduceNLLOutput RooBatchComputeClass::reduceNLL(hipStream_t *stream, RooSpan<const double> probas,
                                                RooSpan<const double> weightSpan, RooSpan<const double> weights,
                                                double weightSum, RooSpan<const double> binVolumes)
{
   ReduceNLLOutput out;
   DeviceArray<double> devOut{gridSize};
   double tmp = 0.0;

   if (weightSpan.size() == 1) {
      nllSumMultiBlock<<<gridSize, blockSize, 0, *stream>>>(probas.data(), probas.size(), devOut.data());
      sumMultiBlock<<<1, blockSize, 0, *stream>>>(devOut.data(), gridSize, devOut.data());
      devOut.copyBack(&tmp, 1);
      tmp *= weightSpan[0];
   } else {
      nllSumWeightedKernel<<<gridSize, blockSize, 0, *stream>>>(probas.data(), weightSpan.data(), devOut.data(),
                                                                probas.size());
      sumMultiBlock<<<1, blockSize, 0, *stream>>>(devOut.data(), gridSize, devOut.data());
      devOut.copyBack(&tmp, 1);
   }

   out.nllSum.Add(tmp);
   return out;
}

/// Static object to trigger the constructor which overwrites the dispatch pointer.
static RooBatchComputeClass computeObj;

/** Construct a Batches object
\param output The array where the computation results are stored.
\param nEvents The number of events to be processed.
\param vars A std::vector containing pointers to the variables involved in the computation.
\param extraArgs An optional std::vector containing extra double values that may participate in the computation.
For every scalar parameter a `Batch` object inside the `Batches` object is set accordingly;
a data member of type double gets assigned the scalar value. This way, when the cuda kernel
is launched this scalar value gets copied automatically and thus no call to hipMemcpy is needed **/
Batches::Batches(RestrictArr output, size_t nEvents, const VarVector &vars, ArgVector &extraArgs, double *)
   : _nEvents(nEvents), _nBatches(vars.size()), _nExtraArgs(extraArgs.size()), _output(output)
{
   if (vars.size() > maxParams) {
      throw std::runtime_error(std::string("Size of vars is ") + std::to_string(vars.size()) +
                               ", which is larger than maxParams = " + std::to_string(maxParams) + "!");
   }
   if (extraArgs.size() > maxExtraArgs) {
      throw std::runtime_error(std::string("Size of extraArgs is ") + std::to_string(extraArgs.size()) +
                               ", which is larger than maxExtraArgs = " + std::to_string(maxExtraArgs) + "!");
   }

   for (int i = 0; i < vars.size(); i++) {
      const RooSpan<const double> &span = vars[i];
      size_t size = span.size();
      if (size == 1)
         _arrays[i].set(span[0], nullptr, false);
      else
         _arrays[i].set(0.0, span.data(), true);
   }
   std::copy(extraArgs.cbegin(), extraArgs.cend(), _extraArgs);
}

} // End namespace RF_ARCH
} // End namespace RooBatchCompute
