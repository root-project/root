/*
 * Project: RooFit
 * Authors:
 *   Emmanouil Michalainas, CERN, September 2020
 *
 * Copyright (c) 2021, CERN
 *
 * Redistribution and use in source and binary forms,
 * with or without modification, are permitted according to the terms
 * listed in LICENSE (http://roofit.sourceforge.net/license.txt)
 */

#include "RooBatchCompute.h"
#include "Batches.h"

#include "TError.h"

#include <thrust/reduce.h>

#ifdef __HIPCC__
#define ERRCHECK(err) __checkCudaErrors((err), __func__, __FILE__, __LINE__)
inline static void __checkCudaErrors(hipError_t error, std::string func, std::string file, int line)
{
   if (error != hipSuccess) {
      Fatal((func + "(), " + file + ":" + std::to_string(line)).c_str(), "%s", hipGetErrorString(error));
      throw std::bad_alloc();
   }
}
#endif

namespace RooBatchCompute {
namespace RF_ARCH {

std::vector<void (*)(Batches)> getFunctions();

class RooBatchComputeClass : public RooBatchComputeInterface {
private:
   const std::vector<void (*)(Batches)> _computeFunctions;

public:
   RooBatchComputeClass() : _computeFunctions(getFunctions())
   {
      dispatchCUDA = this; // Set the dispatch pointer to this instance of the library upon loading
   }

   void init()
   {
      hipError_t err = hipSetDevice(0);
      if (err == hipSuccess)
         hipFree(nullptr);
      else {
         dispatchCUDA = nullptr;
         Error("RbcClass::init()", hipGetErrorString(err));
      }
   }

   void compute(Computer computer, RestrictArr output, size_t nEvents, const DataMap &varData, const VarVector &vars,
                const ArgVector &extraArgs) override
   {
      Batches batches(output, nEvents, varData, vars, extraArgs);
      _computeFunctions[computer]<<<128, 512>>>(batches);
   }
   double sumReduce(InputArr input, size_t n) override { return thrust::reduce(thrust::device, input, input + n, 0.0); }

   // cuda functions
   virtual void *hipMalloc(size_t nBytes)
   {
      void *ret;
      ERRCHECK(::hipMalloc(&ret, nBytes));
      return ret;
   }
   virtual void hipFree(void *ptr) { ERRCHECK(::hipFree(ptr)); }
   virtual void *hipHostMalloc(size_t nBytes)
   {
      void *ret;
      ERRCHECK(::hipHostMalloc(&ret, nBytes));
      return ret;
   }
   virtual void hipHostFree(void *ptr) { ERRCHECK(::hipHostFree(ptr)); }
   virtual hipEvent_t *newCudaEvent(bool forTiming)
   {
      auto ret = new hipEvent_t;
      ERRCHECK(hipEventCreateWithFlags(ret, forTiming ? 0 : hipEventDisableTiming));
      return ret;
   }
   virtual void deleteCudaEvent(hipEvent_t *event)
   {
      ERRCHECK(hipEventDestroy(*event));
      delete event;
   }
   virtual void hipEventRecord(hipEvent_t *event, hipStream_t *stream)
   {
      ERRCHECK(::hipEventRecord(*event, *stream));
   }
   virtual hipStream_t *newCudaStream()
   {
      auto ret = new hipStream_t;
      ERRCHECK(hipStreamCreate(ret));
      return ret;
   }
   virtual void deleteCudaStream(hipStream_t *stream)
   {
      ERRCHECK(hipStreamDestroy(*stream));
      delete stream;
   }
   virtual bool streamIsActive(hipStream_t *stream)
   {
      hipError_t err = hipStreamQuery(*stream);
      if (err == hipErrorNotReady)
         return true;
      else if (err == hipSuccess)
         return false;
      ERRCHECK(err);
      return false;
   }
   virtual void hipStreamWaitEvent(hipStream_t *stream, hipEvent_t *event)
   {
      ERRCHECK(::hipStreamWaitEvent(*stream, *event));
   }
   virtual float hipEventElapsedTime(hipEvent_t *begin, hipEvent_t *end)
   {
      float ret;
      ERRCHECK(::hipEventElapsedTime(&ret, *begin, *end));
      return ret;
   }
   void memcpyToCUDA(void *dest, const void *src, size_t nBytes, hipStream_t *stream) override
   {
      if (stream)
         ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyHostToDevice, *stream));
      else
         ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyHostToDevice));
   }
   void memcpyToCPU(void *dest, const void *src, size_t nBytes, hipStream_t *stream) override
   {
      if (stream)
         ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyDeviceToHost, *stream));
      else
         ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToHost));
   }
}; // End class RooBatchComputeClass

/// Static object to trigger the constructor which overwrites the dispatch pointer.
static RooBatchComputeClass computeObj;

Batches::Batches(RestrictArr output, size_t nEvents, const DataMap &varData, const VarVector &vars,
                 const ArgVector &extraArgs, double[maxParams][bufferSize])
   : _nEvents(nEvents), _nBatches(vars.size()), _nExtraArgs(extraArgs.size()), _output(output)
{
   for (int i = 0; i < vars.size(); i++) {
      const RooSpan<const double> &span = varData.at(vars[i]);
      size_t size = span.size();
      if (size == 1)
         _arrays[i].set(span[0], nullptr, false);
      else
         _arrays[i].set(0.0, span.data(), true);
   }
   std::copy(extraArgs.cbegin(), extraArgs.cend(), _extraArgs);
}

} // End namespace RF_ARCH
} // End namespace RooBatchCompute
