/*
 * Project: RooFit
 * Authors:
 *   Emmanouil Michalainas, CERN, September 2020
 *
 * Copyright (c) 2021, CERN
 *
 * Redistribution and use in source and binary forms,
 * with or without modification, are permitted according to the terms
 * listed in LICENSE (http://roofit.sourceforge.net/license.txt)
 */

#include "RooBatchCompute.h"
#include "Batches.h"

#include "TError.h"

#include <thrust/reduce.h>

namespace RooBatchCompute {
namespace RF_ARCH {

std::vector<void (*)(Batches)> getFunctions();

class RooBatchComputeClass : public RooBatchComputeInterface {
private:
   const std::vector<void (*)(Batches)> computeFunctions;

public:
   RooBatchComputeClass() : computeFunctions(getFunctions())
   {
      dispatchCUDA = this; // Set the dispatch pointer to this instance of the library upon loading
   }

   void init()
   {
      hipError_t err = hipSetDevice(0);
      if (err == hipSuccess)
         hipFree(nullptr);
      else {
         dispatchCUDA = nullptr;
         Error("RbcClass::init()", hipGetErrorString(err));
      }
   }

   void compute(Computer computer, RestrictArr output, size_t nEvents, const DataMap &varData, const VarVector &vars,
                const ArgVector &extraArgs) override
   {
      Batches batches(output, nEvents, varData, vars, extraArgs);
      computeFunctions[computer]<<<128, 512>>>(batches);
   }

   double sumReduce(InputArr input, size_t n) override { return thrust::reduce(thrust::device, input, input + n, 0.0); }

   void *malloc(size_t size) override
   {
      void *ret = nullptr;
      hipError_t error = hipMalloc(&ret, size);
      if (error != hipSuccess) {
         Fatal((std::string(__func__) + "(), " + __FILE__ + ":" + std::to_string(__LINE__)).c_str(), "%s",
               hipGetErrorString(error));
         throw std::bad_alloc();
      } else
         return ret;
   }

   void free(void *ptr) override { hipFree(ptr); }

   void memcpyToGPU(void *dest, const void *src, size_t n) { hipMemcpy(dest, src, n, hipMemcpyHostToDevice); }

   void memcpyToCPU(void *dest, const void *src, size_t n) { hipMemcpy(dest, src, n, hipMemcpyDeviceToHost); }
}; // End class RooBatchComputeClass

/// Static object to trigger the constructor which overwrites the dispatch pointer.
static RooBatchComputeClass computeObj;

Batches::Batches(RestrictArr _output, size_t _nEvents, const DataMap &varData, const VarVector &vars,
                 const ArgVector &_extraArgs, double[maxParams][bufferSize])
   : nEvents(_nEvents), nBatches(vars.size()), nExtraArgs(_extraArgs.size()), output(_output)
{
   for (int i = 0; i < vars.size(); i++) {
      const RooSpan<const double> &span = varData.at(vars[i]);
      size_t size = span.size();
      if (size == 1)
         arrays[i].set(span[0], nullptr, false);
      else
         arrays[i].set(0.0, span.data(), true);
   }
   std::copy(_extraArgs.cbegin(), _extraArgs.cend(), extraArgs);
}

} // End namespace RF_ARCH
} // End namespace RooBatchCompute
