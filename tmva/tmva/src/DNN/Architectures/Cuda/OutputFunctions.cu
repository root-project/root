// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 11/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

////////////////////////////////////////////////////////////////
// Explicit instantiation of the Reference architecture class //
// template for Double_t scalar types.                        //
////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"
#include "Kernels.cuh"

namespace TMVA
{
namespace DNN
{

template<typename AFloat>
void TCuda<AFloat>::Sigmoid(TCudaMatrix<AFloat> & B,
                            const TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Sigmoid<<<gridDims, blockDims, 0, s>>>(B.GetDataPointer(),
                                                             A.GetDataPointer(),
                                                             (int) A.GetNrows(),
                                                             (int) A.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Softmax(TCudaMatrix<AFloat> & B,
                            const TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims1D();
   dim3 gridDims  = TDevice::GridDims1D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Softmax<<<gridDims, blockDims, 0, s>>>(B.GetDataPointer(),
                                                             A.GetDataPointer(),
                                                             (int) A.GetNrows(),
                                                             (int) A.GetNcols());
   B.SetComputeStream(s);
}

} // namespace DNN
} // namespace TMVA
