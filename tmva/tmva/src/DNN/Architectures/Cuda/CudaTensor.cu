// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

/////////////////////////////////////////////
// Implementation of the TCudaTensor class. //
/////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda/CudaTensor.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"

#include <algorithm>
#include <cassert>
#include <iostream>

namespace TMVA {
namespace DNN  {


// Static members.
//____________________________________________________________________________
#ifdef R__HAS_CUDNN
template<typename AFloat>
std::vector<hipdnnHandle_t> TCudaTensor<AFloat>::fCudnnHandle(1);
template<typename AFloat>
hipdnnDataType_t          TCudaTensor<AFloat>::fDataType         = HIPDNN_DATA_FLOAT;
#endif

template<typename AFloat>
std::vector<int>         TCudaTensor<AFloat>::fInstances(1,0);

/// This information is needed for the multi-dimensional indexing. See here:
/// https://en.wikipedia.org/wiki/Row-_and_column-major_order
/// https://docs.scipy.org/doc/numpy/reference/generated/numpy.ndarray.strides.html
template<typename AFloat>
std::vector<std::size_t> TCudaTensor<AFloat>::ComputeStridesFromShape(const std::vector<std::size_t> &shape,
   bool rowmajorLayout)
{
   const auto size = shape.size();
   std::vector<std::size_t> strides(size);
   if (rowmajorLayout)  {
      for (std::size_t i = 0; i < size; i++) {
         if (i == 0) {
            strides[size - 1 - i] = 1;
         } else {
            strides[size - 1 - i] = strides[size - 1 - i + 1] * shape[size - 1 - i + 1];
         }
      }
   } else  {
      for (std::size_t i = 0; i < size; i++) {
         if (i == 0) {
            strides[i] = 1;
         } else {
            strides[i] = strides[i - 1] * shape[i - 1];
         }
      }
   }
   return strides;
}

// Constructors.
//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor()
    : fShape(), fStrides(), fNDim(0), fSize(0), fElementBuffer(), fStreamIndx(0), fTensorDescriptor(nullptr)
{
   //InitializeCuda();
}


//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(const std::vector<size_t> & shape,
                                 TCudaTensor::MemoryLayout layout,
                                 int device, int streamIndx)
    : fShape(shape), fStrides(shape.size()), fNDim(shape.size()), fDevice(device), fStreamIndx(streamIndx),
      fTensorDescriptor(nullptr), fMemoryLayout(layout)
{
   fStrides = ComputeStridesFromShape(fShape, layout==MemoryLayout::RowMajor);

   fSize = (layout==MemoryLayout::RowMajor) ? fStrides.front()*fShape.front() :
                                              fStrides.back()*fShape.back();

   // create a new buffer in this case
   fElementBuffer = TCudaDeviceBuffer<AFloat>(fSize, 0);
   // need to initialize Cuda when creating a new Cuda Buffer (e.g. create Tensor descriptor)
   InitializeCuda();
}

//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(const AFloat * host_data, const std::vector<size_t> & shape,
                                 TCudaTensor::MemoryLayout layout,
                                 int device, int streamIndx)
   : TCudaTensor(shape, layout, device, streamIndx)
{
   // do I need to allocate this buffer ????
   // is not a mem leak
   // AFloat * buffer = new AFloat[fSize];
   // size_t index = 0;
   // for (size_t j = 0; j < fSize; ++j) {
   //       buffer[j] = static_cast<AFloat>(host_data[j]);
   //    }
   // }

   hipMemcpy(fElementBuffer, host_data, fSize * sizeof(AFloat),
              hipMemcpyHostToDevice);

   // no need to initialize cuda. Done in the other constructor that is called before
   //InitializeCuda();
}

//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(TCudaDeviceBuffer<AFloat> buffer,
                                 const std::vector<size_t> & shape,
                                 TMVA::Experimental::MemoryLayout layout,
                                 int device, int streamIndx)
   : fNDim(shape.size()), fElementBuffer(buffer), fShape(shape), fStrides( shape.size()), fDevice(device),
     fStreamIndx(streamIndx), fTensorDescriptor(nullptr), fMemoryLayout(layout)
{
   // constructor from an existing buffer . Buffer size must contain given size
   fStrides = ComputeStridesFromShape(fShape, layout==MemoryLayout::RowMajor);

   fSize = (layout==MemoryLayout::RowMajor) ? fStrides.front()*fShape.front() :
                                              fStrides.back()*fShape.back();
   R__ASSERT(fSize <= buffer.GetSize());

   // need to Initialize Cuda in case device buffer was created separatly
   InitializeCuda();
}

//____________________________________________________________________________
//FIXME: Go to shared_ptr implementation of instance tracking
// template <typename AFloat>
// TCudaTensor<AFloat>::TCudaTensor(const TCudaTensor<AFloat>& oldTensor) :
//    TCudaTensor(oldTensor.fShape, oldTensor.fMemoryLayout, oldTensor.fDevice, oldTensor.fStreamIndx)
// {
//    // No deep copy
//    fStrides       = oldTensor.fStrides;
//    fElementBuffer = oldTensor.fElementBuffer;

//    std::cout << "calling copy constructor of TCuda tensor" << std::endl;

//    InitializeCuda();
// }

//____________________________________________________________________________
template <typename AFloat>
TCudaTensor<AFloat>::TCudaTensor(const TCudaMatrix<AFloat>& matrix, size_t dim) :
   TCudaTensor( matrix.GetDeviceBuffer(), {matrix.GetNrows(), matrix.GetNcols()}, MemoryLayout::ColumnMajor)
{
   // No deep copy
   if (dim > 2) {
      // change shape from (nrows,ncols) to (nrows,ncols,1,1)
      // this works onlt for coolum major layout since this is same of TCudaMatrix
      fShape.insert(fShape.end(), dim-2, 1);
      fStrides.insert(fStrides.end(),dim-2,fSize);
      fNDim = dim;
      // need to reset tensor descriptor since we are changing the shape
      SetTensorDescriptor();
   }
}



template<typename AFloat>
TCudaTensor<AFloat>::operator TMatrixT<AFloat>() const
{
   // this should work only for size 2 or 4 tensors
   if (GetLayout() == MemoryLayout::ColumnMajor &&
       (fNDim == 2 || (fNDim == 3 && GetFirstSize() == 1)) ) {
//         return TCudaMatrix<AFloat>(fElementBuffer, GetHSize(), GetWSize());
      TCudaMatrix<AFloat> temp = GetMatrix();
      return temp;
   }
   // we can convert directy to TMatrix
   // assert(fNDim <= 4);
   // size_t nRows = fShape[0]*fShape[1];
   // size_t nCols = fShape[2];
   // if (fNDim == 4) nCols*= fShape[3];

   if (GetLayout() == MemoryLayout::RowMajor) {

     // This assume that tensor D1, D2, D3,D4 is converted in   D1 , D2*D3*D4
     TMatrixT<AFloat> hostMatrix( GetNrows(), GetNcols() );
     hipMemcpy(hostMatrix.GetMatrixArray(), fElementBuffer, fSize * sizeof(AFloat),
           hipMemcpyDeviceToHost);
     return hostMatrix;

   }
   // else in case of column major tensor we need to transpose(this is what is done in TCudaMatrix)
   // Here we assume that D1, D2, D3 is converted in   a matrix  (D3, D1*D2)
   TMatrixT<AFloat> hostMatrix( GetNcols(), GetNrows() );
   hipMemcpy(hostMatrix.GetMatrixArray(), fElementBuffer, fSize * sizeof(AFloat),
              hipMemcpyDeviceToHost);
   return hostMatrix.T();  // return transpose matrix

}
#ifdef R__HAS_CUDNN
//____________________________________________________________________________
template <typename AFloat>
TCudaTensor<AFloat>::~TCudaTensor()
{
   if (fTensorDescriptor && fTensorDescriptor.use_count() == 1 ) {
      // //std::cout << "Destroy tensor descriptor for shape ";
      // for (int ii = 0; ii < fNDim; ++ii)
      //    std::cout << fShape[ii] << ",";
      // std::cout << std::endl;
      CUDNNCHECK(hipdnnDestroyTensorDescriptor(fTensorDescriptor->fCudnnDesc));

      fInstances[fStreamIndx]--;

         // When all tensors in a streamIndx are destroyed, release cudnn resources
      if (fInstances[fStreamIndx] <= 0) {
         std::cout << "All Cuda tensors are -released - destroy cudnn handle " << fInstances[fStreamIndx] << std::endl;
         CUDNNCHECK(hipdnnDestroy(fCudnnHandle[fStreamIndx]));
      }

   }
   //std::cout << "Tensor descriptor destroyed - instances are " << fInstances[fStreamIndx] << std::endl;

}
//____________________________________________________________________________
template <typename AFloat>
void TCudaTensor<AFloat>::InitializeCuda()
{
   // descriptor is needed for Cuddn tensor that are rowmajor
   if (!fTensorDescriptor && fSize > 0 && fNDim >= 2) {


      // if ((fInstances[fStreamIndx] < 4 && fInstances[fStreamIndx] > -4) || fInstances[fStreamIndx]%1000 == 0) {
      //    std::cout << " stream index " << fStreamIndx << " instances " << fInstances[fStreamIndx] << std::endl;
      //    PrintShape();
      // }


      // Also check whether a new streamIndx has been opened
      if (fInstances.size() - 1 < fStreamIndx) {
         // If need to resize once, need probably to resize more often
         fInstances.resize(2 * fStreamIndx + 1, 0);
         fCudnnHandle.resize(2 * fStreamIndx + 1, nullptr);
         }
      if (fInstances[fStreamIndx] == 0) {
         std::cout << "TCudaTensor::create cudnn handle ! " << std::endl;
         CUDNNCHECK(hipdnnCreate(&fCudnnHandle[fStreamIndx]));
         // CUDNNCHECK(hipdnnSetStream(fCudnnHandle[fStreamIndx], fElementBuffer.GetComputeStream()));

         // hipblasCreate(&fCublasHandle);
         // CUDACHECK(hipMalloc(& fDeviceReturn, sizeof(AFloat)));
         // CUDACHECK(hipMalloc(& fCurandStates, TDevice::NThreads(*this)));
      }
      // if (TDevice::NThreads(*this) > (int) fNCurandStates) {
      //     fNCurandStates = TDevice::NThreads(*this);
      //     if (fCurandStates) {
      //         hipFree(fCurandStates);
      //     }
      //     hipMalloc(&fCurandStates, TDevice::NThreads(*this) * sizeof(hiprandState_t));
      //     InitializeCurandStates();
      // }

      // Prevent template specialization of entire class
      if (std::is_same<AFloat, double>::value) {
         fDataType = HIPDNN_DATA_DOUBLE;
      } else if (std::is_same<AFloat, float>::value) {
         fDataType = HIPDNN_DATA_FLOAT;
      }

      // create tensor descriptor
      fTensorDescriptor = std::make_shared<TensorDescriptor>();
      // std::cout << "create tensor  descriptor ! " << std::endl;
      CUDNNCHECK(hipdnnCreateTensorDescriptor(&(fTensorDescriptor->fCudnnDesc)));

      // we increment instances when we create the descriptor
      fInstances[fStreamIndx]++;
   }

   SetTensorDescriptor();

}
template<typename AFloat>
void TCudaTensor<AFloat>::SetTensorDescriptor() {
      if (!fTensorDescriptor) return;
      if (fSize == 0) return;

      // cuDNN NdTensor format has a minsize of 4 tensor dimensions
      // 4D tensor is more performant on lower dimensions and supports all folowing operations
      // is this really true ???
      if (fNDim == 4 || fNDim > 1 && fMemoryLayout == MemoryLayout::ColumnMajor || fNDim == 2) {
         // pad cudnn tensor column major with extra elements (these are used in the convolutions)
         Shape_t shape = fShape;

         if (fNDim < 4 && fNDim > 1) {
            //    // add 1 to tensor
            if (fMemoryLayout == MemoryLayout::RowMajor)
               shape.insert(shape.end(), 4 - fNDim, 1);
            else
               shape.insert(shape.begin(), 4 - fNDim, 1);
         }

         if (fMemoryLayout == MemoryLayout::RowMajor) {
            auto status = hipdnnSetTensor4dDescriptor(fTensorDescriptor->fCudnnDesc,
                                                     HIPDNN_TENSOR_NCHW, // Layout of the tensor in memory
                                                     fDataType,
                                                     (int)shape[0],  // batch size
                                                     (int)shape[1],  // no. channels
                                                     (int)shape[2],  // image height
                                                     (int)shape[3]); // image width
            assert(status == HIPDNN_STATUS_SUCCESS);
            CUDNNCHECK(status);
         } else {
            CUDNNCHECK(hipdnnSetTensor4dDescriptor(fTensorDescriptor->fCudnnDesc,
                                                  HIPDNN_TENSOR_NCHW, // Layout of the tensor in memory
                                                  fDataType,
                                                  (int)shape[3],   // batch size
                                                  (int)shape[2],   // no. channels
                                                  (int)shape[1],   // image height
                                                  (int)shape[0])); // image width
         }

         // Some operations in cudnn may not work with this tensor description
         // do not support tensors with dims < 1
      } else if (fNDim >2  || fNDim > 4) {
         // these are used in the RNN layers

         // seems to work for 3d tensor with row major (case of RNN tensors)
         // rnn wnats 3d tensors but it does not work for 2d tensors
         std::vector<int> shape(fShape.begin(), fShape.end());
         std::vector<int> strides(fStrides.begin(), fStrides.end());
         auto status = hipdnnSetTensorNdDescriptor(fTensorDescriptor->fCudnnDesc, fDataType, (int)fNDim, shape.data(),
                                                  strides.data());
         assert(status == HIPDNN_STATUS_SUCCESS);
         CUDNNCHECK(status);
      }

#ifdef NDEBUG
      size_t tensorSize;
      CUDNNCHECK(cudnnGetTensorSizeInBytes(fTensorDescriptor->fCudnnDesc, &tensorSize));
      assert(fSize == tensorSize/sizeof(AFloat));

        //    int n,c,h,w = 0;
   // int s1,s2,s3,s4 = 0;
   // hipdnnDataType_t  dataType;
   // hipdnnGetTensor4dDescriptor( fTensorDescriptor, &dataType,&n,&c,&h,&w,&s1,&s2,&s3,&s4 );
   // std::vector<size_t>  shape_input = {n,c,h,w};
   // assert (shape_input == GetShape());

#endif


}
#else // case ROOT has not Cudnn (add dummy implementations)
//____________________________________________________________________________
template <typename AFloat>
TCudaTensor<AFloat>::~TCudaTensor()
{}
//____________________________________________________________________________
template <typename AFloat>
void TCudaTensor<AFloat>::InitializeCuda()
{}
//____________________________________________________________________________
template<typename AFloat>
void TCudaTensor<AFloat>::SetTensorDescriptor()
{}

#endif

//____________________________________________________________________________
template<typename AFloat>
void TCudaTensor<AFloat>::InitializeCurandStates()
{
   // dim3 blockDims = TDevice::BlockDims2D();
   // dim3 gridDims  = TDevice::GridDims2D(*this);
   // CurandInitializationKernel<<<gridDims, blockDims>>>(time(nullptr), fCurandStates);
}

template<typename AFloat>
void TCudaTensor<AFloat>::Print(const char * name, bool truncate) const
{
      //TCudaBuffer<AFloat> hostBuffer (fSize);
      //fElementBuffer.CopyTo(hostBuffer);
    #if 0
      AFloat hostBuffer[fSize];

      hipMemcpy(hostBuffer, fElementBuffer, fSize * sizeof(AFloat),
                 hipMemcpyDeviceToHost);

      for (size_t i = 0; i < fSize; i++) std::cout << hostBuffer[i] << "  ";
   #endif
   PrintShape(name);
   size_t n = fSize;
   if (n > 10 && truncate) n = 10;
   std::cout << "Data : { ";
   for (size_t i = 0; i < n; ++i ) {
      AFloat * elementPointer = fElementBuffer + i;
      std::cout << AFloat( TCudaDeviceReference<AFloat>(elementPointer) );
      if (i < n-1) std::cout << " , ";
   }
   if (n < fSize) std::cout << "............   } ";
   std::cout << " } " << std::endl;
}
template<typename AFloat>
void TCudaTensor<AFloat>::PrintShape(const char * name) const
{
      std::string memlayout = (GetLayout() == MemoryLayout::RowMajor) ? "RowMajor" : "ColMajor";
      std::cout << name << " shape : { ";
      for (size_t i = 0; i < fNDim-1; ++i )
         std::cout << fShape[i] << " , ";
      std::cout << fShape.back() << " } " << " Layout : " << memlayout << std::endl;
}
#if 0
// Conversion to RTensor
//____________________________________________________________________________
template<typename AFloat>
TCudaTensor<AFloat>::operator Experimental::RTensor<AFloat>() const
{
   std::vector<size_t> shape(fNDims, fNDims + fDim)

   Experimental::RTensor<AFloat> hostTensor( shape)

   AFloat * buffer = new AFloat[fSize];
   hipMemcpy(buffer, fElementBuffer, fSize * sizeof(AFloat),
              hipMemcpyDeviceToHost);

   int index = 0;
   for (int j = 0; j < fSize; j++) {
         hostTensor.GetData()[j] = static_cast<AFloat>(buffer[j]);
      }
   }

   delete[] buffer;
   return hostTensor;
}
#endif
// Explicit Instantiations.

template class TCudaTensor<float>;
template class TCudaTensor<double>;

} // namespace DNN
} // namespace TMVA
